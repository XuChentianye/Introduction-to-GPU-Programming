#include "hip/hip_runtime.h"
#include<stdio.h>
#include"error_check.h"
#include"gpu_timer.h"

#define DTYPE float
#define DTYPE_OUTPUT_FORMAT "%f "

#define H 1024
#define W 977
#define BLOCK_SIZE 32


void transpose_CPU(DTYPE *input, DTYPE *output, int num_rows, int num_cols)
{
	for(int row_idx=0; row_idx<num_rows; row_idx++)
	{
		for(int col_idx=0; col_idx<num_cols; col_idx++)
		{
			output[col_idx*num_rows+row_idx] = input[row_idx*num_cols+col_idx];
		}
	}
}

__global__ void kernel_transpose_serial(DTYPE *input, DTYPE *output, int num_rows, int num_cols)
{
	int input_width = num_cols;
	int output_width = num_rows;
	for(int row_idx=0; row_idx<num_rows; row_idx++)
	{
		for(int col_idx=0; col_idx<num_cols; col_idx++)
		{
			output[col_idx*output_width+row_idx] = input[row_idx*input_width+col_idx];
		}
	}
}

__global__ void kernel_transpose_per_row(DTYPE *input, DTYPE *output, int num_rows, int num_cols)
{
	int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int input_width = num_cols;
	int output_width = num_rows;
	for(int row_idx = 0; row_idx<num_rows; row_idx++)
	{
		if(row_idx<num_rows && col_idx<num_cols)
		{
			output[col_idx*output_width+row_idx] = input[row_idx*input_width+col_idx];
		}
	}
}

__global__ void kernel_transpose_per_element(DTYPE *input, DTYPE *output, int num_rows, int num_cols)
{
	int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

	int input_width = num_cols;
	int output_width = num_rows;

	if(row_idx<num_rows && col_idx<num_cols)
	{
		output[col_idx*output_width+row_idx] = input[row_idx*input_width+col_idx];
	}
}

/* 
* Todo:
* Implement the kernel function while satisfying the following requirements*
* 1.1 Utilizing shared memory to achieve coalesced memory access to both input and output matrices *
*/

__global__ void kernel_transpose_per_element_tiled(DTYPE *input, DTYPE *output, int num_rows, int num_cols)
{
	int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int input_width = num_cols;
	int output_width = num_rows;

	__shared__ DTYPE tile[BLOCK_SIZE][BLOCK_SIZE];
	tile[threadIdx.y][threadIdx.x] = input[row_idx*input_width+col_idx];
	__syncthreads();

	// compute the target block index
	int blockIdx_x_target = blockIdx.y;
	int blockIdx_y_target = blockIdx.x;
	int col_idx_target = blockIdx_x_target * blockDim.y + threadIdx.x;
	int row_idx_target = blockIdx_y_target * blockDim.x + threadIdx.y;
	output[row_idx_target*output_width+col_idx_target] = tile[threadIdx.x][threadIdx.y];
}

/* 
* Todo:
* Implement the kernel function while satisfying the following requirements*
* 2.1 Utilizing shared memory to achieve coalesced memory access to both input and output matrices *
* 2.2 Avoid bank conflicts * 
*/
__global__ void kernel_transpose_per_element_tiled_no_bank_conflicts(DTYPE *input, DTYPE *output, int num_rows, int num_cols)
{
	int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int input_width = num_cols;
	int output_width = num_rows;

	__shared__ DTYPE tile[BLOCK_SIZE+1][BLOCK_SIZE];
	int thy = threadIdx.y + ( (threadIdx.x + threadIdx.y < BLOCK_SIZE) ? 0 : 1 );
	int thx = (threadIdx.x + threadIdx.y) % BLOCK_SIZE;
	tile[thy][thx] = input[row_idx*input_width+col_idx];
	__syncthreads();

	// compute the target block index
	int blockIdx_x_target = blockIdx.y;
	int blockIdx_y_target = blockIdx.x;
	int col_idx_target = blockIdx_x_target * blockDim.y + threadIdx.x;
	int row_idx_target = blockIdx_y_target * blockDim.x + threadIdx.y;
	int thx_ = threadIdx.x + ( (threadIdx.x + threadIdx.y < BLOCK_SIZE) ? 0 : 1 );
	int thy_ = (threadIdx.x + threadIdx.y) % BLOCK_SIZE;

	output[row_idx_target*output_width+col_idx_target] = tile[thx_][thy_];

}


void init_data(DTYPE *arr, int n)
{
	for(int i=0; i<n; i++){
		arr[i] = (DTYPE)(i+1);
	}
}

int compare_matrices(DTYPE *input, DTYPE *ref, int num_rows, int num_cols)
{
	for(int row_idx=0; row_idx<num_rows; row_idx++)
	{
		for(int col_idx=0; col_idx<num_cols; col_idx++)
		{
			if(abs(ref[row_idx*num_cols+col_idx]-input[row_idx*num_cols+col_idx])>1e-3)
			{
				printf("Error:%f at (%d, %d)\n", abs(ref[row_idx*num_cols+col_idx]-input[row_idx*num_cols+col_idx]), row_idx, col_idx);
				return 0;
			}
		}
	}
	return 1;
}

int main() 
{
	int numBytes = H*W*sizeof(DTYPE);
	DTYPE *data_input = (DTYPE *)malloc(numBytes);
	DTYPE *data_output = (DTYPE *)malloc(numBytes);
	DTYPE *data_result = (DTYPE *)malloc(numBytes);

	init_data(data_input, H*W);
	transpose_CPU(data_input, data_result, H, W);

	DTYPE *d_in, *d_out;
	hipMalloc((void **)&d_in, numBytes);
	hipMalloc((void **)&d_out, numBytes);
	hipMemcpy(d_in, data_input, numBytes, hipMemcpyHostToDevice);
	GpuTimer timer;

	/* 
	* 1. matrix transpose serial *
	*/
	timer.Start();
	kernel_transpose_serial<<<1, 1>>>(d_in, d_out, H, W);
	timer.Stop();
	hipMemcpy(data_output, d_out, numBytes, hipMemcpyDeviceToHost);
	printf("\nTime cost (serial):%g ms. Veryifying results...%s\n",
			timer.Elapsed(), compare_matrices(data_output, data_result, H, W)?"Passed":"Failed");
	memset(data_output, 0, numBytes);
	hipMemset(d_out, 0, numBytes);

	
	/* 
	* 2. matrix transpose per row *
	*/
	timer.Start();
	kernel_transpose_per_row<<<1, W>>>(d_in, d_out, H, W);
	timer.Stop();
	hipMemcpy(data_output, d_out, numBytes, hipMemcpyDeviceToHost);
	printf("\nTime cost (per row):%g ms. Veryifying results...%s\n",
			timer.Elapsed(), compare_matrices(data_output, data_result, H, W)?"Passed":"Failed");
	memset(data_output, 0, numBytes);
	hipMemset(d_out, 0, numBytes);


	/* 
	* 3. matrix transpose per element *
	*/
	timer.Start();
	dim3 blocks((W-1)/BLOCK_SIZE+1, (H-1)/BLOCK_SIZE+1);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	kernel_transpose_per_element<<<blocks, threads>>>(d_in, d_out, H, W);
	timer.Stop();
	hipMemcpy(data_output, d_out, numBytes, hipMemcpyDeviceToHost);
	printf("\nTime cost (per element):%g ms. Veryifying results...%s\n",
			timer.Elapsed(), compare_matrices(data_output, data_result, H, W)?"Passed":"Failed");
	memset(data_output, 0, numBytes);
	hipMemset(d_out, 0, numBytes);


	/* 
	* 4. matrix transpose tiled with shared memory *
	*/
	timer.Start();
	kernel_transpose_per_element_tiled<<<blocks, threads>>>(d_in, d_out, H, W);
	timer.Stop();
	hipMemcpy(data_output, d_out, numBytes, hipMemcpyDeviceToHost);
	printf("\nTime cost (tiled with shared memory):%g ms. Veryifying results...%s\n",
			timer.Elapsed(), compare_matrices(data_output, data_result, H, W)?"Passed":"Failed");
	memset(data_output, 0, numBytes);
	hipMemset(d_out, 0, numBytes);


	/* 
	* 5. matrix transpose tiled without bank conflicts *
	*/
	timer.Start();
	kernel_transpose_per_element_tiled_no_bank_conflicts<<<blocks, threads>>>(d_in, d_out, H, W);
	timer.Stop();
	hipMemcpy(data_output, d_out, numBytes, hipMemcpyDeviceToHost);
	printf("\nTime cost (tiled without bank conflicts):%g ms. Veryifying results...%s\n",
			timer.Elapsed(), compare_matrices(data_output, data_result, H, W)?"Passed":"Failed");



	free(data_input);
	free(data_output);
	free(data_result);
	hipFree(d_in);
	hipFree(d_out);

    return 0;
}
