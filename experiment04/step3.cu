#include "hip/hip_runtime.h"
#include"error_check.h"
#include <stdio.h>
#define PWD 3

__global__ void entrypt_p(char *in, char *out)
{
    int idx = threadIdx.x;
    out[idx] = in[idx] + PWD;
}

int main()
{
    char A[] = "Hello, world!";
    int memSize = strlen(A)*sizeof(char);
    int strLen = strlen(A);
    char *B = (char *)malloc(memSize);
    printf("Input: \n");
    for(int i=0; i<strLen; i++){printf("%c ", A[i]);}

    //ToDo
    char *d_A=NULL;
    char *d_B=NULL;
    CHECK(hipMalloc((void **)&d_B, memSize));
    CHECK(hipMalloc((void **)&d_A, memSize));
    CHECK(hipMemcpy(d_A, A, memSize, hipMemcpyHostToDevice));
    entrypt_p<<<1,strlen(A)>>>(d_A, d_B);
    CHECK(hipMemcpy(B, d_B, memSize, hipMemcpyDeviceToHost));
    printf("\nOutput: \n");
    for(int i=0; i<strLen; i++){printf("%c ", B[i]);}
    printf("\n");
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    return 0;
}
