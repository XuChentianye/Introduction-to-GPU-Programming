
#include <hip/hip_runtime.h>
#include<stdio.h>

int main() {

    hipDeviceProp_t prop;

    int device_id = 0;
    printf("\nGet properties from device #%d:\n", device_id);
    hipError_t error_code = hipGetDeviceProperties(&prop, device_id);
    if(error_code==hipSuccess)
    {
        printf("CUDA API successed!\n");
    }
    else if(error_code==hipErrorInvalidDevice)
    {
        printf("Invalid Device! code:%d \n", error_code);
    }

    device_id = 1;
    printf("\nGet properties from device #%d:\n", device_id);
    error_code = hipGetDeviceProperties(&prop, device_id);
    if(error_code==hipSuccess)
    {
        printf("CUDA API successed!\n");
    }
    else if(error_code==hipErrorInvalidDevice)
    {
        printf("Invalid Device! code:%d \n", error_code);
        printf("line:%d in %s\n", __LINE__, __FILE__);
    }
    return 0;
}
