#include "hip/hip_runtime.h"
#include"error_check.h"
#include<stdio.h>

__global__ void func1(int x)
{
    int tid = threadIdx.x;
    printf("thread: %d, parameter:%d  \n", tid, x);
}

__global__ void func2(int *x)
{
    int tid = threadIdx.x;
    printf("thread: %d, parameter:%d  \n", tid, *x);
}

__global__ void func3(int x[], int n)
{
    int tid = threadIdx.x;
    printf("thread: %d, parameter:%d  \n", tid, x[tid]);
}

int main() {
	int a = 6, temp = 7;
	int *b, *tem = &temp;
	int n = 3;
	int tempc[3] = {0,1,2};
	int *c;

	hipMalloc((void**)&b, sizeof(int));
	CHECK(hipGetLastError());
	hipMemcpy(b, tem, sizeof(int), hipMemcpyHostToDevice);
	CHECK(hipGetLastError());
	hipMalloc((void**)&c, 3*sizeof(int));
	CHECK(hipGetLastError());
	hipMemcpy(c, tempc, 3*sizeof(int), hipMemcpyHostToDevice);
	CHECK(hipGetLastError());

	func1<<<1, 1>>>(a);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	printf("\n");

	func2<<<1, 1>>>(b);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	printf("\n");
	func3<<<1, 1>>>(c, n);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	printf("\n");
	CHECK(hipDeviceReset());

    return 0;
}
