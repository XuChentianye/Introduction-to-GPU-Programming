
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void hello(){
    printf("*");
}

int main() {
    hipError_t error_code;
    hello<<<-1, 1>>>();

    error_code = hipGetLastError();
    if(error_code!=hipSuccess){
        printf("\n");
        printf("line:%d in %s\n", __LINE__, __FILE__);
        printf("Error needs to be handled!\n");
        printf("Error code:%d \n", error_code);
        printf("Error string:%s \n", hipGetErrorString(error_code));
    }

    hello<<<1, 1025>>>();
    error_code = hipGetLastError();
    if(error_code!=hipSuccess){
        printf("\n");
        printf("line:%d in %s\n", __LINE__, __FILE__);
        printf("Error needs to be handled!\n");
        printf("Error code:%d \n", error_code);
        printf("Error string:%s \n", hipGetErrorString(error_code));
    }

    error_code = hipDeviceSynchronize();
    if(error_code!=hipSuccess){
        printf("\n");
        printf("line:%d in %s\n", __LINE__, __FILE__);
        printf("Error needs to be handled!\n");
        printf("Error code:%d \n", error_code);
        printf("Error string:%s \n", hipGetErrorString(error_code));
    }

    error_code  = hipDeviceReset();
    if(error_code!=hipSuccess){
        printf("\n");
        printf("line:%d in %s\n", __LINE__, __FILE__);
        printf("Error needs to be handled!\n");
        printf("Error code:%d \n", error_code);
        printf("Error string:%s \n", hipGetErrorString(error_code));
    }


    return 0;
}
