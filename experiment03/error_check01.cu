
#include <hip/hip_runtime.h>
#include<stdio.h>

int main() {
    hipError_t error_code = hipDeviceReset();
    printf("returned error code:%d \n", error_code); // a cudaError_t variable can be regarded as a integer
    printf("cudaSuccess:%d, error_code==cudaSuccess:%d \n", hipSuccess, hipSuccess==error_code);
    if(error_code==hipSuccess)
    {
        printf("CUDA API successed!\n");
    }
    else
    {
        printf("Error needs to be handled! code:%d \n", error_code);
    }
    return 0;
}
