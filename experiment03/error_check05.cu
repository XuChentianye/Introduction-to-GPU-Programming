#include "hip/hip_runtime.h"
#include"error_check.h"
#include<stdio.h>
__global__ void hello(){
    printf("Hello World!\n");
}

int main() {
    hello<<<-1, 1>>>();
    CHECK(hipGetLastError());

    hello<<<1, 1025>>>();
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());   

    CHECK(hipDeviceReset());
    return 0;
}
